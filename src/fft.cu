/*
 *  Copyright 2016 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

/*! \file fft.cu
 *  \brief This file wraps cufft functionality into the Bifrost C++ API.
 */
#include <hipfft/hipfft.h>
#if BF_CUDA_ENABLED
    #include "cuda/stream.hpp"
    #include <hip/hip_runtime_api.h>
    #define FFT_FORWARD HIPFFT_FORWARD
    #define FFT_INVERSE HIPFFT_BACKWARD
    #define FFT_C2C HIPFFT_C2C 
    #define FFT_R2C HIPFFT_R2C 
    #define FFT_C2R HIPFFT_C2R 
#endif
#include <bifrost/common.h>
#include <bifrost/ring.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#define BF_MAX_DIM 3

/// Defines a single atom of data to be passed to a function.
typedef struct BFarray_ {
    /*! The data pointer can point towards any type of data, 
     *  so long as there is a corresponding definition in dtype. 
     *  This data should be an ndim array, which every element of
     *  type dtype.
     */
    void* data;
    /*! Where this data is located in memory.
     *  Used to ensure that operations called are localized within
     *  that space, such as a CUDA funciton operating on device
     *  memory.
     */
    BFspace space;
    unsigned dtype;
    int ndim;
    BFsize shape[BF_MAX_DIM];
    BFsize strides[BF_MAX_DIM];
} BFarray;

typedef float BFcomplex[2];
typedef float BFreal;

/*! \brief Calls a 1 dimensional CUDA FFT.
 *
 *  @param[in] input_data 
 *  \parblock
 *  Pointer to one dimensional array
 *  of untransformed data. 
 *
 *  This input_data must be signed 32 floating point.
 *  \endparblock
 *  @param[in] nelements Number of elements in input array
 *  @param[in] direction (FFT_FORWARD/FFT_INVERSE)
 *  @param[out] output_data 
 *  \parblock
 *  Pointer to one dimensional array to hold ouput.
 * 
 *  Must be pre-allocated.
 *  \endparblock
 *  \returns Whether or not computation was a success.
 */
BFstatus bfFFTC2C1d(
    void** input_data, void** output_data, 
    BFsize nelements, int direction)
{
    hipfftComplex* idata = *((hipfftComplex**)input_data);
    hipfftComplex* odata = *((hipfftComplex**)output_data);
    hipfftHandle plan;
    hipfftPlan1d(&plan, nelements, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, idata, odata, direction);
    return BF_STATUS_SUCCESS;
}

/*! \brief Calls a 2 dimensional CUDA FFT.
 *
 *  @param[in] input_data 
 *  \parblock
 *  Pointer to two dimensional array
 *  of untransformed data. 
 *
 *  This input_data must be signed 32 floating point.
 *  \endparblock
 *  @param[in] nelements_x Number of elements in input 
 *  along x-dimension in input array
 *  @param[in] nelements_y Number of elements in input 
 *  along y-dimension in input array
 *  @param[in] direction (FFT_FORWARD/FFT_INVERSE)
 *  @param[out] output_data 
 *  \parblock
 *  Pointer to two dimensional array to hold ouput.
 * 
 *  Must be pre-allocated.
 *  \endparblock
 *  \returns Whether or not computation was a success.
 */
BFstatus bfFFTC2C2d(
    void** input_data, void** output_data, 
    BFsize nelements_x, BFsize nelements_y, 
    int direction)
{
    hipfftComplex* idata = *((hipfftComplex**)input_data);
    hipfftComplex* odata = *((hipfftComplex**)output_data);
    hipfftHandle plan;
    hipfftPlan2d(&plan, nelements_x, nelements_y, HIPFFT_C2C);
    hipfftExecC2C(plan, idata, odata, direction);
    return BF_STATUS_SUCCESS;
}

/*! \brief Calls a 1 dimensional CUDA FFT on real input.
 *
 *  @param[in] input_data 
 *  \parblock
 *  Pointer to one dimensional array
 *  of untransformed data. 
 *
 *  This input_data must be signed 32 floating point.
 *  \endparblock
 *  @param[in] nelements Number of elements in input array
 *  @param[out] output_data 
 *  \parblock
 *  Pointer to one dimensional array to hold ouput.
 * 
 *  Must be pre-allocated.
 *  \endparblock
 *  \returns Whether or not computation was a success.
 */
BFstatus bfFFTR2C1d(
    void** input_data, void** output_data, 
    BFsize nelements)
{
    hipfftReal* idata = *((hipfftReal**)input_data);
    hipfftComplex* odata = *((hipfftComplex**)output_data);
    hipfftHandle plan;
    hipfftPlan1d(&plan, nelements, HIPFFT_R2C, 1);
    hipfftExecR2C(plan, idata, odata);
    return hipGetLastError();
}

/*! \brief Calls a 2 dimensional CUDA FFT on real
 *  input
 *
 *  @param[in] input_data 
 *  \parblock
 *  Pointer to two dimensional array
 *  of untransformed data. 
 *
 *  This input_data must be signed 32 floating point.
 *  \endparblock
 *  @param[in] nelements_x Number of elements in input 
 *  along x-dimension in input array
 *  @param[in] nelements_y Number of elements in input 
 *  along y-dimension in input array
 *  @param[out] output_data 
 *  \parblock
 *  Pointer to two dimensional array to hold ouput.
 * 
 *  Must be pre-allocated.
 *  \endparblock
 *  \returns Whether or not computation was a success.
 */
BFstatus bfFFTR2C2d(
    void** input_data, void** output_data, 
    BFsize nelements_x, BFsize nelements_y)
{
    hipfftReal* idata = *((hipfftReal**)input_data);
    hipfftComplex* odata = *((hipfftComplex**)output_data);
    hipfftHandle plan;
    hipfftPlan2d(&plan, nelements_x, nelements_y, HIPFFT_R2C);
    hipfftExecR2C(plan, idata, odata);
    return hipGetLastError();
}

/*! \brief Calls a complex FFT function based on 
 *          specifications in BFarrays
 *
 *  @param[in] input - pointer to BFarray that contains 
 *  data to be transformed, with description of that
 *  data
 *  @param[in] direction (FFT_FORWARD/FFT_INVERSE)
 *  @param[out] output - pointer to BFarray which will 
 *  contain the transformed data
 *  \returns Whether or not computation was a success.
 */
BFstatus bfFFT(
    BFarray *input, BFarray *output, int direction)
{
    // TODO: Move plan here.
    // TODO: Use planMany instead of plan1d.
    // TODO: Set up BF dtype enum.
    // TODO: Make this function support type conversion
    // TODO: Enable multiple GPU support.
    hipfftHandle fftPlan; 
    if (input->dtype == 0)
    {
        if (input->ndim == 1)
            return bfFFTR2C1d(
                (void**)&(input->data), (void**)&(output->data),
                input->shape[0]);
        else if (input->ndim == 2)
            return bfFFTR2C2d(
                (void**)&(input->data), (void**)&(output->data),
                input->shape[0], input->shape[1]);
    }
    if (input->ndim == 1)
        return bfFFTC2C1d(
            (void**)&(input->data), (void**)&(output->data),
            input->shape[0], direction);
    else if(input->ndim == 2)
        return bfFFTC2C2d(
            (void**)&(input->data), (void**)&(output->data),
            input->shape[0], input->shape[1], direction);
    return BF_STATUS_INTERNAL_ERROR;
}

