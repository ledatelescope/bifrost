/*
 *  Copyright 2016 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

/*! \file fft.cu
 *  \brief This file wraps cufft functionality into the Bifrost C++ API.
 */

#include <hipfft/hipfft.h>
#if BF_CUDA_ENABLED
    #include "cuda/stream.hpp"
    #include <hip/hip_runtime_api.h>
    #define FFT_FORWARD HIPFFT_FORWARD
    #define FFT_INVERSE HIPFFT_BACKWARD
    #define FFT_C2C HIPFFT_C2C 
    #define FFT_R2C HIPFFT_R2C 
    #define FFT_C2R HIPFFT_C2R 
#endif
#include <bifrost/common.h>
#include <bifrost/ring.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#define BF_MAX_DIM 3

/// Defines a single atom of data to be passed to a function.
typedef struct BFarray_ {
    /*! The data pointer can point towards any type of data, 
     *  so long as there is a corresponding definition in dtype. 
     *  This data should be an ndim array, which every element of
     *  type dtype.
     */
    void* data;
    /*! Where this data is located in memory.
     *  Used to ensure that operations called are localized within
     *  that space, such as a CUDA funciton operating on device
     *  memory.
     */
    BFspace space;
    unsigned dtype;
    int ndim;
    BFsize shape[BF_MAX_DIM];
    BFsize strides[BF_MAX_DIM];
} BFarray;

typedef float BFcomplex[2];
typedef float BFreal;

/*! \brief Calls a 1 dimensional CUDA FFT.
 *
 *  inputs:
 *  input_data - a pointer to one dimensional array
 *       of untransformed data
 *  nelements - number of elements in in input array
 *  dtype - datatype of input array. Assumed complex.
 *  space - where data is located
 *  outputs:
 *  output_data - pointer to one dimensional array 
 *       of transformed data
 *  Returns whether or not the operation was a success.
 */
BFstatus bfFFTC2C1d(
    void** input_data, void** output_data, 
    BFsize nelements, unsigned dtype,
    BFspace space, int direction)
{
    hipfftComplex* idata = *((hipfftComplex**)input_data);
    hipfftComplex* odata = *((hipfftComplex**)output_data);
    hipfftHandle plan;
    hipfftPlan1d(&plan, nelements, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, idata, odata, direction);
    return BF_STATUS_SUCCESS;
}

/*! \brief Calls a 2 dimensional CUDA FFT.
 *
 *  inputs:
 *  input_data - a pointer to two dimensional array
 *       of untransformed data
 *  nelements_x - number of elements in input array
 *       along x dimension
 *  nelements_y - number of elements in input array
 *       along y dimension
 *  dtype - datatype of input array. Assumed complex.
 *  space - where data is located
 *  outputs:
 *  output_data - pointer to one dimensional array 
 *       of transformed data
 *  Returns whether or not the operation was a success.
 */
BFstatus bfFFTC2C2d(
    void** input_data, void** output_data, 
    BFsize nelements_x, BFsize nelements_y, 
    unsigned dtype, BFspace space, int direction)
{
    hipfftComplex* idata = *((hipfftComplex**)input_data);
    hipfftComplex* odata = *((hipfftComplex**)output_data);
    hipfftHandle plan;
    hipfftPlan2d(&plan, nelements_x, nelements_y, HIPFFT_C2C);
    hipfftExecC2C(plan, idata, odata, direction);
    return BF_STATUS_SUCCESS;
}

/*! \brief Calls a 1 dimensional real-real CUDA FFT
 *
 *  inputs:
 *  input_data - a pointer to two dimensional array
 *       of untransformed data
 *  nelements - number of elements in input array
 *  dtype - datatype of input array. Assumed complex.
 *  space - where data is located
 *  outputs:
 *  output_data - pointer to one dimensional array 
 *       of transformed data
 *  Returns whether or not the operation was a success.
 */
BFstatus bfFFTR2C1d(
    void** input_data, void** output_data, 
    BFsize nelements, unsigned dtype, 
    BFspace space)
{
    hipfftReal* idata = *((hipfftReal**)input_data);
    hipfftComplex* odata = *((hipfftComplex**)output_data);
    hipfftHandle plan;
    hipfftPlan1d(&plan, nelements, HIPFFT_R2C, 1);
    hipfftExecR2C(plan, idata, odata);
    return hipGetLastError();
}

/*! \brief Calls a 2 dimensional real-real CUDA FFT
 *
 *  inputs:
 *  input_data - a pointer to two dimensional array
 *       of untransformed data
 *  nelements_x - number of elements in input array
 *       along x dimension
 *  nelements_y - number of elements in input array
 *       along y dimension
 *  dtype - datatype of input array. Assumed complex.
 *  stride - number of bytes for each element
 *  space - where data is located
 *  outputs:
 *  output_data - pointer to one dimensional array 
 *       of transformed data
 *  Returns whether or not the operation was a success.
 */
BFstatus bfFFTR2C2d(
    void** input_data, void** output_data, 
    BFsize nelements_x, BFsize nelements_y,
    unsigned dtype, BFspace space)
{
    hipfftReal* idata = *((hipfftReal**)input_data);
    hipfftComplex* odata = *((hipfftComplex**)output_data);
    hipfftHandle plan;
    hipfftPlan2d(&plan, nelements_x, nelements_y, HIPFFT_R2C);
    hipfftExecR2C(plan, idata, odata);
    return hipGetLastError();
}

/*! \brief Calls a complex FFT function based on 
 *          specifications in BFarrays
 *
 *  inputs:
 *  input - pointer to BFarray that contains data to be
 *       transformed
 *  outputs:
 *  output - pointer to BFarray which will contain the
 *       transformed data
 *  Returns whether or not the operation was a success.
 */
BFstatus bfFFT(
    BFarray *input, BFarray *output, int direction)
{
    // TODO: Move plan here.
    // TODO: Make user pass FFT_R2C
    // TODO: Provide same functionality as in cufft_nyquist_packed.cu
    // TODO: Set Ben's callbacks.
    // TODO: Use planMany instead of plan1d.
    // TODO: Set up BF dtype variable.
    if (input->dtype == 0)
    {
        if (input->ndim == 1)
            return bfFFTR2C1d(
                (void**)&(input->data), (void**)&(output->data),
                input->shape[0], input->dtype,
                input->space);
        else if (input->ndim == 2)
            return bfFFTR2C2d(
                (void**)&(input->data), (void**)&(output->data),
                input->shape[0], input->shape[1],
                input->dtype, input->space);
    }
    if (input->ndim == 1)
        return bfFFTC2C1d(
            (void**)&(input->data), (void**)&(output->data),
            input->shape[0], input->dtype,
            input->space, direction);
    else if(input->ndim == 2)
        return bfFFTC2C2d(
            (void**)&(input->data), (void**)&(output->data),
            input->shape[0], input->shape[1], input->dtype,
            input->space, direction);
    return BF_STATUS_INTERNAL_ERROR;
}

void test_bffft_real_2d()
{
    BFarray my_data;
    BFarray out_data;
    BFreal set_data[3][2] = 
        {{1,2},{2,3},{3,4}};
    BFreal** some_data;
    BFcomplex* odata;
    hipMalloc((void**)&some_data, sizeof(BFreal)*6);
    hipMalloc((void**)&odata, sizeof(BFcomplex)*6);
    hipMemcpy(
        some_data, set_data, 
        sizeof(BFreal)*6, hipMemcpyHostToDevice);
    my_data.data = some_data;
    my_data.space = BF_SPACE_CUDA;
    my_data.shape[0] = 3;
    my_data.shape[1] = 2;
    my_data.dtype = 0;
    my_data.ndim = 2;
    my_data.strides[0] = 2*sizeof(BFreal);
    my_data.strides[1] = sizeof(BFreal);
    out_data = my_data;
    out_data.data = odata;
    out_data.dtype = 1;
    out_data.strides[0] = 2*sizeof(BFcomplex);
    out_data.strides[1] = sizeof(BFcomplex);
    if (bfFFT(&my_data, &out_data, FFT_FORWARD) != BF_STATUS_SUCCESS)
    {
        printf("bfFFT failed!\n");
        return; 
    }
    hipfftComplex localdata[3][2] = {};
    hipMemcpy(
        localdata, (hipfftComplex*)out_data.data, 
        sizeof(hipfftComplex)*6, hipMemcpyDeviceToHost);
    for(int i = 0; i < 3; i++)
    {
        for (int j = 0; j < 2; j++)
        {
            printf(
                "%f+I%f\n",
                hipCrealf(localdata[i][j]),
                hipCimagf(localdata[i][j]));
        }
    }
    return;
}
void test_bffft_real()
{
    BFarray my_data;
    BFarray out_data;
    BFreal set_data[4] = {1,3,6,2.5134};
    BFreal* some_data;
    BFcomplex* odata;
    hipMalloc((void**)&some_data, sizeof(BFreal)*5);
    hipMalloc((void**)&odata, sizeof(BFcomplex)*3);
    hipMemcpy(
        some_data, set_data, 
        sizeof(BFreal)*4, hipMemcpyHostToDevice);
    my_data.data = some_data;
    my_data.space = BF_SPACE_CUDA;
    my_data.shape[0] = 4;
    my_data.dtype = 0;
    my_data.ndim = 1;
    my_data.strides[0] = sizeof(BFreal);
    out_data = my_data;
    out_data.data = odata;
    out_data.dtype = 1;
    out_data.strides[0] = sizeof(BFcomplex);
    if (bfFFT(&my_data, &out_data, FFT_FORWARD) != BF_STATUS_SUCCESS)
    {
        printf("bfFFT failed!\n");
        return; 
    }
    hipfftComplex localdata[3] = {};
    hipMemcpy(
        localdata, (hipfftComplex*)out_data.data, 
        sizeof(hipfftComplex)*3, hipMemcpyDeviceToHost);
    for(int i = 0; i < 3; i++)
        printf("%f+I%f\n",hipCrealf(localdata[i]),hipCimagf(localdata[i]));
    return;
}

void test_bffft_2d()
{
    BFarray my_data;
    BFcomplex set_data[3][3] = 
        {{{5,1},{0,0},{100,0}},
        {{5,1},{30,0},{100,0}},
        {{30,0},{0,0},{10,1}}};
    BFcomplex** some_data;
    hipMalloc((void**)&some_data, sizeof(BFcomplex)*9);
    hipMemcpy(
        some_data, set_data, 
        sizeof(BFcomplex)*9, hipMemcpyHostToDevice);
    my_data.data = some_data;
    my_data.space = BF_SPACE_CUDA;
    my_data.shape[0] = 3;
    my_data.shape[1] = 3;
    my_data.dtype = 1;
    my_data.ndim = 2;
    my_data.strides[0] = 3*sizeof(BFcomplex);
    my_data.strides[1] = sizeof(BFcomplex);
    if (bfFFT(&my_data, &my_data, FFT_FORWARD) != BF_STATUS_SUCCESS)
    {
        printf("bfFFT failed!\n");
        return; 
    }
    hipfftComplex localdata[3][3]={};
    hipMemcpy(
        localdata, (hipfftComplex**)my_data.data, 
        sizeof(hipfftComplex)*9, hipMemcpyDeviceToHost);
    for(int i = 0; i < 3; i++)
    {
        for (int j = 0; j < 3; j++)
                printf("%f\n",hipCrealf(localdata[i][j]));
    }
    //print successfully fft'd data.
    return;
}

void test_bffft_1d()
{
    BFarray my_data;
    BFcomplex set_data[5] = {{0,0},{30,0},{100,0},{30,0},{-5,0}};
    BFcomplex* some_data;
    hipMalloc((void**)&some_data, sizeof(BFcomplex)*5);
    hipMemcpy(some_data, set_data, sizeof(BFcomplex)*5, hipMemcpyHostToDevice);
    my_data.data = some_data;
    my_data.space = BF_SPACE_CUDA;
    my_data.shape[0] = 5;
    my_data.dtype = 1;
    my_data.ndim = 1;
    my_data.strides[0] = sizeof(BFcomplex);
    bfFFT(&my_data, &my_data, FFT_FORWARD);
    hipfftComplex localdata[5]={};
    hipMemcpy(localdata, (hipfftComplex*)my_data.data, sizeof(hipfftComplex)*5, hipMemcpyDeviceToHost);
    for(int i = 0; i < 5; i++)
        printf("%f+I%f\n",hipCrealf(localdata[i]),hipCimagf(localdata[i]));
    //print successfully fft'd data.
}


int main()
{
    printf("Running...\n");
    //test_bffft_1d();
    //test_bffft_2d();
    //test_bffft_real();
    test_bffft_real_2d();
    printf("Done\n");
    return 0;
}
