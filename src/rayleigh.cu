#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, The Bifrost Authors. All rights reserved.
 * Copyright (c) 2022, The University of New Mexico. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * * Redistributions of source code must retain the above copyright
 *   notice, this list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimer in the
 *   documentation and/or other materials provided with the distribution.
 * * Neither the name of The Bifrost Authors nor the names of its
 *   contributors may be used to endorse or promote products derived
 *   from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <bifrost/rayleigh.h>
#include "assert.hpp"
#include "utils.hpp"
#include "workspace.hpp"
#include "cuda.hpp"
#include "trace.hpp"
#include "Complex.hpp"

//#include <limits>

#include <hip/hip_math_constants.h> // For HIP_NAN_F
#include <thrust/device_vector.h>
#include <hiprand.h>

#include <vector>
#include <map>
#include <string>

// HACK TESTING
#include <iostream>
using std::cout;
using std::endl;

#define BF_POOL_SIZE 268435456

// cuRAND API errors - from 
static const char *curandGetErrorString(hiprandStatus_t error)
{
    switch (error)
    {
        case HIPRAND_STATUS_SUCCESS:
            return "HIPRAND_STATUS_SUCCESS";

        case HIPRAND_STATUS_VERSION_MISMATCH:
            return "HIPRAND_STATUS_VERSION_MISMATCH";

        case HIPRAND_STATUS_NOT_INITIALIZED:
            return "HIPRAND_STATUS_NOT_INITIALIZED";

        case HIPRAND_STATUS_ALLOCATION_FAILED:
            return "HIPRAND_STATUS_ALLOCATION_FAILED";

        case HIPRAND_STATUS_TYPE_ERROR:
            return "HIPRAND_STATUS_TYPE_ERROR";

        case HIPRAND_STATUS_OUT_OF_RANGE:
            return "HIPRAND_STATUS_OUT_OF_RANGE";

        case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
            return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";

        case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
            return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";

        case HIPRAND_STATUS_LAUNCH_FAILURE:
            return "HIPRAND_STATUS_LAUNCH_FAILURE";

        case HIPRAND_STATUS_PREEXISTING_FAILURE:
            return "HIPRAND_STATUS_PREEXISTING_FAILURE";

        case HIPRAND_STATUS_INITIALIZATION_FAILED:
            return "HIPRAND_STATUS_INITIALIZATION_FAILED";

        case HIPRAND_STATUS_ARCH_MISMATCH:
            return "HIPRAND_STATUS_ARCH_MISMATCH";

        case HIPRAND_STATUS_INTERNAL_ERROR:
            return "HIPRAND_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

#define BF_CHECK_CURAND_EXCEPTION(call, err) \
	do { \
		hiprandStatus_t cuda_ret = call; \
		if( cuda_ret != HIPRAND_STATUS_SUCCESS ) { \
			BF_DEBUG_PRINT(curandGetErrorString(cuda_ret)); \
		} \
		/*BF_ASSERT(cuda_ret == hipSuccess, err);*/ \
		BF_ASSERT_EXCEPTION(cuda_ret == HIPRAND_STATUS_SUCCESS, err); \
	} while(0)

#define BF_CHECK_CURAND(call, err) \
	do { \
		hiprandStatus_t cuda_ret = call; \
		if( cuda_ret != HIPRAND_STATUS_SUCCESS ) { \
			BF_DEBUG_PRINT(curandGetErrorString(cuda_ret)); \
		} \
		BF_ASSERT(cuda_ret == CURAND_STATUS_SUCCES, err); \
	} while(0)

template<typename InType>
__global__ void flagger_kernel(unsigned int               ntime,
                               unsigned int               nantpol,
                               float                      alpha,
                               unsigned int               clip_sigmas,
                               float                      max_flag_frac,
                               float*                     state,
                               const float* __restrict__  pool,
                               unsigned int*              flags,
                               const InType* __restrict__ d_in,
                               InType* __restrict__       d_out) {
	int a = threadIdx.x + blockIdx.x*blockDim.x;
	
  int r = a;
  if( r > BF_POOL_SIZE ) r %= BF_POOL_SIZE;
  
	int t, count, bad_count;
  float power, mean;
	if( a < nantpol ) {
    mean = 0.0;
    count = bad_count = 0;
    
		for(t=0; t<ntime; t++) {
      power  = d_in[t*nantpol*2 + a*2 + 0]*d_in[t*nantpol*2 + a*2 + 0];
      power += d_in[t*nantpol*2 + a*2 + 1]*d_in[t*nantpol*2 + a*2 + 1];
      
      if( power >= (clip_sigmas*sqrt(4/HIP_PI_F-1)*state[a]) && state[a] != 0 ) {
        d_out[t*nantpol*2 + a*2 + 0] = pool[r++] * sqrt(2/HIP_PI_F)*state[a];
        if( r > BF_POOL_SIZE ) r = 0;
        d_out[t*nantpol*2 + a*2 + 1] = pool[r++] * sqrt(2/HIP_PI_F)*state[a];
        if( r > BF_POOL_SIZE ) r = 0;
				
        bad_count++;
      } else {
        d_out[t*nantpol*2 + a*2 + 0] = d_in[t*nantpol*2 + a*2 + 0];
        d_out[t*nantpol*2 + a*2 + 1] = d_in[t*nantpol*2 + a*2 + 1];
				
				mean += power;
	      count++;
			}
		}
    
    mean /= count;
    if( bad_count < (count*max_flag_frac)) {
      state[a] = alpha*mean + (1-alpha)*state[a];
    } else {
      atomicAdd(flags, 1);
    }
	}
}

template<typename InType>
inline void launch_flagger_kernel(unsigned int  ntime, 
                                  unsigned int  nantpol,
                                  float         alpha,
                                  unsigned int  clip_sigmas,
                                  float         max_flag_frac,
                                  float*        state,
                                  float*        pool,
                                  BFsize*       flags,
                                  InType*       d_in,
                                  InType*       d_out,
                                  hipStream_t  stream=0) {
	//cout << "LAUNCH for " << nelement << endl;
	dim3 block(std::min(256u, nantpol), nantpol/std::min(256u, nantpol));
	int first = std::min((nantpol-1)/block.x+1, 65535u);
	dim3 grid(first, 1u, 1u);

  /*
	cout << "  Block size is " << block.x << " by " << block.y << endl;
	cout << "  Grid  size is " << grid.x << " by " << grid.y << " by " << grid.z << endl;
	cout << "  Maximum size is " << block.y*grid.y*grid.z << endl;
	*/
	
	BF_CHECK_CUDA_EXCEPTION(hipMemsetAsync(flags,
	                                        0,
	                                        sizeof(BFsize),
	                                        stream),
                         	BF_STATUS_MEM_OP_FAILED );
														 					 
	void* args[] = {&ntime, 
	                &nantpol,
	                &alpha,
                  &clip_sigmas,
	                &max_flag_frac,
	                &state,
                  &pool,
                  &flags,
	                &d_in,
	                &d_out};
	BF_CHECK_CUDA_EXCEPTION(hipLaunchKernel((void*)flagger_kernel<InType>,
	                                         grid, block,
	                                         &args[0], 0, stream),
	                        BF_STATUS_INTERNAL_ERROR);
}

class BFrayleigh_impl {
	typedef int          IType;
	typedef unsigned int UType;
	typedef double       FType;
public: // HACK WAR for what looks like a bug in the CUDA 7.0 compiler
	typedef float  DType;
private:
	UType     _nantpol;
  float     _alpha;
  UType     _clip_sigmas;
  float     _max_flag_frac;
  float*    _state = NULL;
  float*    _pool = NULL;
	BFsize*   _flags = NULL;
	IType     _plan_stride;
	Workspace _plan_storage;
	// TODO: Use something other than Thrust
	thrust::device_vector<char> _dv_plan_storage;
	hipStream_t _stream;
public:
	BFrayleigh_impl() : _stream(g_cuda_stream) {}
	inline UType nantpol()  const { return _nantpol; }
	void init(UType nantpol, 
            float alpha,
	          UType clip_sigmas,
            float max_flag_frac) {
		BF_TRACE();
		_nantpol       = nantpol;
    _alpha         = alpha;
    _clip_sigmas   = clip_sigmas;
    _max_flag_frac = max_flag_frac;
		
		_state = NULL;
	}
	bool init_plan_storage(void* storage_ptr, BFsize* storage_size) {
		BF_TRACE();
		BF_TRACE_STREAM(_stream);
		enum {
			ALIGNMENT_BYTES = 512,
			ALIGNMENT_ELMTS = ALIGNMENT_BYTES / sizeof(float)
		};
		Workspace workspace(ALIGNMENT_BYTES);
		_plan_stride = round_up(_nantpol, ALIGNMENT_ELMTS);
		workspace.reserve(_nantpol+1, &_state);
    workspace.reserve(BF_POOL_SIZE+1, &_pool);
		workspace.reserve(1, &_flags);
		
		if( storage_size ) {
			if( !storage_ptr ) {
				// Return required storage size
				*storage_size = workspace.size();
				return false;
			} else {
				BF_ASSERT_EXCEPTION(*storage_size >= workspace.size(),
				                    BF_STATUS_INSUFFICIENT_STORAGE);
			}
		} else {
			// Auto-allocate storage
			BF_ASSERT_EXCEPTION(!storage_ptr, BF_STATUS_INVALID_ARGUMENT);
			_dv_plan_storage.resize(workspace.size());
			storage_ptr = thrust::raw_pointer_cast(&_dv_plan_storage[0]);
		}
		workspace.commit(storage_ptr);
		
		this->reset_state();
		return true;
	}
	void reset_state() {
		BF_ASSERT_EXCEPTION(_state != NULL,  BF_STATUS_INVALID_STATE);
		BF_ASSERT_EXCEPTION(_pool != NULL,   BF_STATUS_INVALID_STATE);
		
		BF_CHECK_CUDA_EXCEPTION(hipGetLastError(), BF_STATUS_INTERNAL_ERROR);
		
		// Reset the state
		BF_CHECK_CUDA_EXCEPTION( hipMemsetAsync(_state,
		                                         0,
		                                         sizeof(float)*_nantpol,
		                                         _stream),
		                         BF_STATUS_MEM_OP_FAILED );
		BF_CHECK_CUDA_EXCEPTION( hipStreamSynchronize(_stream),
 		                         BF_STATUS_DEVICE_ERROR );
 		
 		BF_CHECK_CUDA_EXCEPTION(hipGetLastError(), BF_STATUS_INTERNAL_ERROR);
		
    hiprandGenerator_t gen;
    BF_CHECK_CURAND_EXCEPTION(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT),
                              BF_STATUS_DEVICE_ERROR);
    BF_CHECK_CURAND_EXCEPTION(hiprandSetPseudoRandomGeneratorSeed(gen, 102114111115116ULL),
                              BF_STATUS_DEVICE_ERROR);
    BF_CHECK_CURAND_EXCEPTION(hiprandGenerateNormal(gen, _pool, BF_POOL_SIZE, 0.0, 1.0),
                              BF_STATUS_DEVICE_ERROR);
    BF_CHECK_CURAND_EXCEPTION(hiprandDestroyGenerator(gen), BF_STATUS_DEVICE_ERROR);
		
		BF_CHECK_CUDA_EXCEPTION( hipStreamSynchronize(_stream),
		                         BF_STATUS_DEVICE_ERROR );
		
		BF_CHECK_CUDA_EXCEPTION(hipGetLastError(), BF_STATUS_INTERNAL_ERROR);
	}
	void execute(BFarray const* in,
	             BFarray const* out,
               BFsize*        flags) {
		BF_TRACE();
		BF_TRACE_STREAM(_stream);
		BF_ASSERT_EXCEPTION(_state != NULL, BF_STATUS_INVALID_STATE);
		BF_ASSERT_EXCEPTION(out->dtype == BF_DTYPE_CF32 || \
		                    out->dtype == BF_DTYPE_CF64,     BF_STATUS_UNSUPPORTED_DTYPE);
		
		BF_CHECK_CUDA_EXCEPTION(hipGetLastError(), BF_STATUS_INTERNAL_ERROR);
		
#define LAUNCH_FLAGGER_KERNEL(IterType) \
		launch_flagger_kernel(in->shape[0], _nantpol, \
                          _alpha, _clip_sigmas, _max_flag_frac, \
		                      _state, _pool, _flags, \
		                      (IterType)in->data, (IterType)out->data, \
		                      _stream); \
		hipMemcpyAsync(flags, _flags, sizeof(BFsize), hipMemcpyDeviceToHost, _stream);
		
    *flags = 0;
		switch( in->dtype ) {
			case BF_DTYPE_CI8:  LAUNCH_FLAGGER_KERNEL(int8_t*);  break;
			case BF_DTYPE_CI16: LAUNCH_FLAGGER_KERNEL(int16_t*); break;
			case BF_DTYPE_CI32: LAUNCH_FLAGGER_KERNEL(int32_t*); break;
			case BF_DTYPE_CI64: LAUNCH_FLAGGER_KERNEL(int64_t*); break;
			case BF_DTYPE_CF32: LAUNCH_FLAGGER_KERNEL(float*);   break;
			case BF_DTYPE_CF64: LAUNCH_FLAGGER_KERNEL(double*);  break;
			default: BF_ASSERT_EXCEPTION(false, BF_STATUS_UNSUPPORTED_DTYPE);
		}
#undef LAUNCH_FLAGGER_KERNEL
	}
	void set_stream(hipStream_t stream) {
		_stream = stream;
	}
};

BFstatus bfRayleighCreate(BFrayleigh* plan_ptr) {
	BF_TRACE();
	BF_ASSERT(plan_ptr, BF_STATUS_INVALID_POINTER);
	BF_TRY_RETURN_ELSE(*plan_ptr = new BFrayleigh_impl(),
	                   *plan_ptr = 0);
}

BFstatus bfRayleighInit(BFrayleigh plan,
                        BFsize     nantpols,
                        float      alpha,
                        BFsize     clip_sigmas,
                        float      max_flag_frac,
                        BFspace    space,
                        void*      plan_storage,
                        BFsize*    plan_storage_size) {
	BF_TRACE();
	BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
	BF_ASSERT(space_accessible_from(space, BF_SPACE_CUDA),
	          BF_STATUS_UNSUPPORTED_SPACE);
  
  BF_ASSERT((alpha > 0) && (alpha <= 1), BF_STATUS_INVALID_ARGUMENT);
	
	BF_TRY(plan->init(nantpols, alpha, clip_sigmas, max_flag_frac));
	BF_TRY_RETURN(plan->init_plan_storage(plan_storage, plan_storage_size));
}
BFstatus bfRayleighSetStream(BFrayleigh  plan,
                             void const* stream) {
	BF_TRACE();
	BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
	BF_ASSERT(stream, BF_STATUS_INVALID_POINTER);
	BF_TRY_RETURN(plan->set_stream(*(hipStream_t*)stream));
}
BFstatus bfRayleighResetState(BFrayleigh plan) {
	BF_TRY_RETURN(plan->reset_state());
}
BFstatus bfRayleighExecute(BFrayleigh     plan,
                           BFarray const* in,
                           BFarray const* out,
                           BFsize*        flags) {
	BF_TRACE();
	BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
	BF_ASSERT(in,   BF_STATUS_INVALID_POINTER);
	BF_ASSERT(out,  BF_STATUS_INVALID_POINTER);
	BF_ASSERT( in->ndim >= 2,        BF_STATUS_INVALID_SHAPE);
	BF_ASSERT(out->ndim == in->ndim, BF_STATUS_INVALID_SHAPE);
	
	BFarray out_flattened, in_flattened;
	if( in->ndim > 2 ) {
		// Keep the first dim but attempt to flatten all others
		unsigned long keep_dims_mask = 0x1;
		keep_dims_mask |= padded_dims_mask(out);
		keep_dims_mask |= padded_dims_mask(in);
		flatten(out, &out_flattened, keep_dims_mask);
		flatten(in,   &in_flattened, keep_dims_mask);
		out = &out_flattened;
		in  =  &in_flattened;
		BF_ASSERT(in_flattened.ndim == out_flattened.ndim,         BF_STATUS_INTERNAL_ERROR);
		BF_ASSERT(in_flattened.ndim == 2,                          BF_STATUS_UNSUPPORTED_SHAPE);
		BF_ASSERT(in_flattened.shape[1] == out_flattened.shape[1], BF_STATUS_INVALID_SHAPE);
	}
	BF_ASSERT( in->shape[1] == plan->nantpol(), BF_STATUS_INVALID_SHAPE);
	BF_ASSERT(out->shape[1] == in->shape[1],    BF_STATUS_INVALID_SHAPE);
	
	BF_ASSERT(space_accessible_from( in->space, BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
	BF_ASSERT(space_accessible_from(out->space, BF_SPACE_CUDA), BF_STATUS_INVALID_SPACE);
	BF_TRY_RETURN(plan->execute(in, out, flags));
}

BFstatus bfRayleighDestroy(BFrayleigh plan) {
	BF_TRACE();
	BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
	delete plan;
	return BF_STATUS_SUCCESS;
}
